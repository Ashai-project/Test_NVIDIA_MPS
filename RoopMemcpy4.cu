#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define ROOP 3000000
#define SIZE 10000
#define PARA 13
int main(int argc, char **argv)
{
    int **send_buff_h, **send_buff_d, **recv_buff_h, **recv_buff_d;
    hipStream_t *st;
    hipHostMalloc((void **)&recv_buff_h, sizeof(size_t) * PARA, hipHostMallocDefault);
    hipHostMalloc((void **)&send_buff_d, sizeof(size_t) * PARA, hipHostMallocDefault);
    hipHostMalloc((void **)&recv_buff_d, sizeof(size_t) * PARA, hipHostMallocDefault);
    hipHostMalloc((void **)&send_buff_h, sizeof(size_t) * PARA, hipHostMallocDefault);
    hipHostMalloc((void **)&st, sizeof(hipStream_t) * PARA, hipHostMallocDefault);
    // メモリ確保
    for (int i = 0; i < PARA; i++)
    {
        hipMalloc((void **)&send_buff_d[i], sizeof(int) * SIZE);
        hipMalloc((void **)&recv_buff_d[i], sizeof(int) * SIZE);
        hipHostMalloc((void **)&send_buff_h[i], sizeof(int) * SIZE, hipHostMallocDefault);
        hipHostMalloc((void **)&recv_buff_h[i], sizeof(int) * SIZE, hipHostMallocDefault);
        hipStreamCreate(&st[i]);
    }

    // init
    for (int j = 0; j < PARA; j++)
    {
        for (int i = 0; i < SIZE; i++)
        {
            send_buff_h[j][i] = i;
        }
        hipMemcpy(send_buff_d[j], send_buff_h[j], sizeof(int) * SIZE, hipMemcpyHostToDevice);
    }

    // メモリコピー
    auto start = std::chrono::system_clock::now();
    for (int i = 0; i < ROOP; i++)
    {
        for (int j = 0; j < PARA; j++)
        {
            hipMemcpyAsync(recv_buff_d[j], send_buff_d[j], sizeof(int) * SIZE, hipMemcpyDeviceToDevice, st[j]);
        }
    }

    for (int j = 0; j < PARA; j++)
    {
        hipStreamSynchronize(st[j]);
        hipMemcpy(recv_buff_h[j], recv_buff_d[j], sizeof(int) * SIZE, hipMemcpyDeviceToHost);
    }
    auto end = std::chrono::system_clock::now();
    double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "time : " << elapsed << "ms" << std::endl;

    // メモリ解放
    for (int j = 0; j < PARA; j++)
    {
        hipFree(send_buff_d[j]);
        hipFree(recv_buff_d[j]);
        hipHostFree(send_buff_h[j]);
        hipHostFree(recv_buff_h[j]);
    }
    hipHostFree(send_buff_d);
    hipHostFree(recv_buff_d);
    hipHostFree(send_buff_h);
    hipHostFree(recv_buff_h);
}