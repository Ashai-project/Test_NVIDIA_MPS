/**
 * @file 06TestHostAlloc.cu
 * @brief ホストメモリについての知識
 * デバイスからホストメモリの参照は
 * 1.MallocHostのメモリをMemcpyでデバイスに渡す
 * 2.HostAllocで参照可能なホストメモリを確保する
 * が考えられる。
 * これについて、デバイスからの参照が多い場合には2のほうが早い
 * @version 0.1
 * @date 2024-05-16
 *
 * @copyright Copyright (c) 2024
 *
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#define N 1000
#define ROOP 500000

__global__ void add(int *culc_buff, int *buff)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    culc_buff[tid] += buff[tid];
}

int main(int argc, char **argv)
{
    int *send_buff_h, *send_buff_d, *culc_buff, *recv_buff_h;
    // メモリ確保
    hipHostAlloc((void **)&send_buff_h, sizeof(int) * N, hipHostMallocDefault);
    hipHostMalloc((void **)&recv_buff_h, sizeof(int) * N, hipHostMallocDefault);

    for (int i = 0; i < ROOP; i++)
    {
        send_buff_h[i] = 1;
    }
    hipMalloc((void **)&send_buff_d, sizeof(int) * N);
    hipMalloc((void **)&culc_buff, sizeof(int) * N);
    hipMemset(culc_buff, 0, sizeof(int) * N);

    auto start = std::chrono::system_clock::now();
    for (int i = 0; i < ROOP; i++)
    {
        add<<<N, 1>>>(culc_buff, send_buff_h);
    }

    auto end = std::chrono::system_clock::now();
    double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "time : " << elapsed << "ms" << std::endl;

    hipMemcpy(recv_buff_h, culc_buff, sizeof(int) * N, hipMemcpyDeviceToHost);
    std::cout << "value : " << recv_buff_h[0] << std::endl;

    hipFree(send_buff_d);
    hipFree(culc_buff);
    hipHostFree(send_buff_h);
    hipHostFree(recv_buff_h);
}