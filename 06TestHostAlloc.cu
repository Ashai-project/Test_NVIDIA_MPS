
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#define N 1000
#define ROOP 500000

__global__ void add(int *culc_buff, int *buff)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    culc_buff[tid] += buff[tid];
}

int main(int argc, char **argv)
{
    int *send_buff_h, *send_buff_d, *culc_buff, *recv_buff_h;
    // メモリ確保
    hipHostAlloc((void **)&send_buff_h, sizeof(int) * N, hipHostMallocDefault);
    hipHostMalloc((void **)&recv_buff_h, sizeof(int) * N, hipHostMallocDefault);

    for (int i = 0; i < ROOP; i++)
    {
        send_buff_h[i] = 1;
    }
    hipMalloc((void **)&send_buff_d, sizeof(int) * N);
    hipMalloc((void **)&culc_buff, sizeof(int) * N);
    hipMemset(culc_buff, 0, sizeof(int) * N);

    auto start = std::chrono::system_clock::now();
    for (int i = 0; i < ROOP; i++)
    {
        add<<<N, 1>>>(culc_buff, send_buff_h);
    }

    auto end = std::chrono::system_clock::now();
    double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "time : " << elapsed << "ms" << std::endl;

    hipMemcpy(recv_buff_h, culc_buff, sizeof(int) * N, hipMemcpyDeviceToHost);
    std::cout << "value : " << recv_buff_h[0] << std::endl;

    hipFree(send_buff_d);
    hipFree(culc_buff);
    hipHostFree(send_buff_h);
    hipHostFree(recv_buff_h);
}