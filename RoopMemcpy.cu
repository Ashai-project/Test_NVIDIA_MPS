#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define ROOP 1000000
#define SIZE 10000
int main(int argc, char **argv)
{
    int *send_buff_h, *send_buff_d, *recv_buff_h, *recv_buff_d;
    // メモリ確保
    hipMalloc((void **)&send_buff_d, sizeof(int) * SIZE);
    hipMalloc((void **)&recv_buff_d, sizeof(int) * SIZE);
    hipHostMalloc((void **)&send_buff_h, sizeof(int) * SIZE, hipHostMallocDefault);
    hipHostMalloc((void **)&recv_buff_h, sizeof(int) * SIZE, hipHostMallocDefault);

    // init
    for (int i = 0; i < SIZE; i++)
    {
        send_buff_h[i] = i;
    }
    hipMemcpy(send_buff_d, send_buff_h, sizeof(int) * SIZE, hipMemcpyHostToDevice);

    // メモリコピー
    auto start = std::chrono::system_clock::now();
    for (int i = 0; i < ROOP; i++)
    {
        hipMemcpy(recv_buff_d, send_buff_d, sizeof(int) * SIZE, hipMemcpyDeviceToDevice);
    }
    hipMemcpy(recv_buff_h, recv_buff_d, sizeof(int) * SIZE, hipMemcpyDeviceToHost);
    auto end = std::chrono::system_clock::now();
    double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "time : " << elapsed << "ms" << std::endl;
    std::cout << "recv[0] : " << recv_buff_h[0] << std::endl;
    std::cout << "recv[1] : " << recv_buff_h[1] << std::endl;
    std::cout << "recv[2] : " << recv_buff_h[2] << std::endl;

    // メモリ解放
    hipFree(send_buff_d);
    hipFree(recv_buff_d);
    hipHostFree(send_buff_h);
    hipHostFree(recv_buff_h);
}