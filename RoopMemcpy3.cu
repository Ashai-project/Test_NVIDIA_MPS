/**
 * @file RoopMemcpy3.cu
 * @brief ストリームを使用する
 * MPSを使用してもプロセスごとに固有のデフォルトストリームがあるっぽい
 * @version 0.1
 * @date 2024-05-15
 *
 * @copyright Copyright (c) 2024
 *
 */
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define ROOP 3000000
#define SIZE 10000
int main(int argc, char **argv)
{
    int *send_buff_h, *send_buff_d, *recv_buff_h, *recv_buff_d;
    hipStream_t st;
    hipStreamCreate(&st);
    // メモリ確保
    hipMalloc((void **)&send_buff_d, sizeof(int) * SIZE);
    hipMalloc((void **)&recv_buff_d, sizeof(int) * SIZE);
    hipHostMalloc((void **)&send_buff_h, sizeof(int) * SIZE, hipHostMallocDefault);
    hipHostMalloc((void **)&recv_buff_h, sizeof(int) * SIZE, hipHostMallocDefault);

    // init
    for (int i = 0; i < SIZE; i++)
    {
        send_buff_h[i] = i;
    }
    hipMemcpy(send_buff_d, send_buff_h, sizeof(int) * SIZE, hipMemcpyHostToDevice);

    // メモリコピー
    auto start = std::chrono::system_clock::now();
    for (int i = 0; i < ROOP; i++)
    {
        hipMemcpyAsync(recv_buff_d, send_buff_d, sizeof(int) * SIZE, hipMemcpyDeviceToDevice, st);
    }
    hipStreamSynchronize(st);
    hipMemcpy(recv_buff_h, recv_buff_d, sizeof(int) * SIZE, hipMemcpyDeviceToHost);
    auto end = std::chrono::system_clock::now();
    double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "time : " << elapsed << "ms" << std::endl;
    std::cout << "recv[0] : " << recv_buff_h[0] << std::endl;
    std::cout << "recv[1] : " << recv_buff_h[1] << std::endl;
    std::cout << "recv[2] : " << recv_buff_h[2] << std::endl;

    // メモリ解放
    hipFree(send_buff_d);
    hipFree(recv_buff_d);
    hipHostFree(send_buff_h);
    hipHostFree(recv_buff_h);
}