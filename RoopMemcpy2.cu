#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define ROOP 3000000
#define SIZE 10000
#define PARA 13
int main(int argc, char **argv)
{
    int **send_buff_h, **send_buff_d, **recv_buff_h, **recv_buff_d;
    hipHostMalloc((void **)&recv_buff_h, sizeof(size_t), hipHostMallocDefault);
    hipHostMalloc((void **)&send_buff_d, sizeof(size_t), hipHostMallocDefault);
    hipHostMalloc((void **)&recv_buff_d, sizeof(size_t), hipHostMallocDefault);
    hipHostMalloc((void **)&send_buff_h, sizeof(size_t), hipHostMallocDefault);
    // メモリ確保
    for (int i = 0; i < PARA; i++)
    {
        hipMalloc((void **)&send_buff_d[i], sizeof(int) * SIZE);
        hipMalloc((void **)&recv_buff_d[i], sizeof(int) * SIZE);
        hipHostMalloc((void **)&send_buff_h[i], sizeof(int) * SIZE, hipHostMallocDefault);
        hipHostMalloc((void **)&recv_buff_h[i], sizeof(int) * SIZE, hipHostMallocDefault);
    }

    // init
    for (int j = 0; j < PARA; j++)
    {
        for (int i = 0; i < SIZE; i++)
        {
            send_buff_h[j][i] = i;
        }
        hipMemcpy(send_buff_d[j], send_buff_h[j], sizeof(int) * SIZE, hipMemcpyHostToDevice);
    }

    // メモリコピー
    auto start = std::chrono::system_clock::now();
    for (int i = 0; i < ROOP; i++)
    {
        for (int j = 0; j < PARA; j++)
        {
            hipMemcpy(recv_buff_d[j], send_buff_d[j], sizeof(int) * SIZE, hipMemcpyDeviceToDevice);
        }
    }
    for (int j = 0; j < PARA; j++)
    {
        hipMemcpy(recv_buff_h[j], recv_buff_d[j], sizeof(int) * SIZE, hipMemcpyDeviceToHost);
    }
    auto end = std::chrono::system_clock::now();
    double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "time : " << elapsed << "ms" << std::endl;

    // メモリ解放
    for (int j = 0; j < PARA; j++)
    {
        hipFree(send_buff_d[j]);
        hipFree(recv_buff_d[j]);
        hipHostFree(send_buff_h[j]);
        hipHostFree(recv_buff_h[j]);
    }
}